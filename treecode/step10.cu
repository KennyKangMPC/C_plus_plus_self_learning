
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

int   const THREADS = 8;
float const EPS2    = 0.0001;
__device__ __constant__ int constsDevc[2];

__global__ void multipole(float4 *targetGlob, float *multipGlob) {
  float R, R3, R5;
  float3 d;
  float4 target = targetGlob[blockIdx.x * THREADS + threadIdx.x];
  target.w = 0;
  d.x = target.x - multipGlob[0];
  d.y = target.y - multipGlob[1];
  d.z = target.z - multipGlob[2];
  R = rsqrtf(d.x * d.x + d.y * d.y + d.z * d.z);
  R3 = R * R * R;
  R5 = R3 * R * R;
  target.w += multipGlob[ 3] * R;
  target.w += multipGlob[ 4] * (-d.x * R3);
  target.w += multipGlob[ 5] * (-d.y * R3);
  target.w += multipGlob[ 6] * (-d.z * R3);
  target.w += multipGlob[ 7] * (3 * d.x * d.x * R5 - 1 * R3);
  target.w += multipGlob[ 8] * (3 * d.y * d.y * R5 - 1 * R3);
  target.w += multipGlob[ 9] * (3 * d.z * d.z * R5 - 1 * R3);
  target.w += multipGlob[10] * (3 * d.x * d.y * R5);
  target.w += multipGlob[11] * (3 * d.y * d.z * R5);
  target.w += multipGlob[12] * (3 * d.z * d.x * R5);
  targetGlob[blockIdx.x * THREADS + threadIdx.x] = target;
}

int main() {
  int N = 33;
  int Nround = (N  / THREADS + 1) * THREADS;
  int Mround = (13 / THREADS + 1) * THREADS;
  int     constsHost[2] = {N, Nround};
  float4 *sourceHost, *sourceDevc;
  float4 *targetHost, *targetDevc;
  float  *multipHost, *multipDevc;
// Allocate memory on host and device
  sourceHost = (float4*)     malloc( Nround*sizeof(float4) );
  targetHost = (float4*)     malloc( Nround*sizeof(float4) );
  multipHost = (float *)     malloc( Mround*sizeof(float ) );
  hipMalloc(  (void**) &sourceDevc, Nround*sizeof(float4) );
  hipMalloc(  (void**) &targetDevc, Nround*sizeof(float4) );
  hipMalloc(  (void**) &multipDevc, Mround*sizeof(float ) );
// Initialize
  for( int i=0; i<N; i++ ) {
    targetHost[i].x = -rand() / (1. + RAND_MAX);
    targetHost[i].y = -rand() / (1. + RAND_MAX);
    targetHost[i].z = -rand() / (1. + RAND_MAX);
    sourceHost[i].x =  rand() / (1. + RAND_MAX);
    sourceHost[i].y =  rand() / (1. + RAND_MAX);
    sourceHost[i].z =  rand() / (1. + RAND_MAX);
    sourceHost[i].w = 1.0 / N;
  }
  multipHost[0] = multipHost[1] = multipHost[2] = 0.5;
  for( int i=3; i<13; i++ ) multipHost[i] = 0;
  float dx,dy,dz;
  for( int j=0; j<N; j++ ) {
    dx = multipHost[0] - sourceHost[j].x;
    dy = multipHost[1] - sourceHost[j].y;
    dz = multipHost[2] - sourceHost[j].z;
    multipHost[ 3] += sourceHost[j].w;
    multipHost[ 4] += sourceHost[j].w * dx;
    multipHost[ 5] += sourceHost[j].w * dy;
    multipHost[ 6] += sourceHost[j].w * dz;
    multipHost[ 7] += sourceHost[j].w * dx * dx / 2;
    multipHost[ 8] += sourceHost[j].w * dy * dy / 2;
    multipHost[ 9] += sourceHost[j].w * dz * dz / 2;
    multipHost[10] += sourceHost[j].w * dx * dy / 2;
    multipHost[11] += sourceHost[j].w * dy * dz / 2;
    multipHost[12] += sourceHost[j].w * dz * dx / 2;
  }
// Direct summation on device
  hipMemcpyToSymbol(HIP_SYMBOL(constsDevc),constsHost,     2*sizeof(int   ));
  hipMemcpy        (targetDevc,targetHost,Nround*sizeof(float4),hipMemcpyHostToDevice);
  hipMemcpy        (multipDevc,multipHost,Mround*sizeof(float ),hipMemcpyHostToDevice);
  multipole<<< Nround/THREADS, THREADS >>>(targetDevc,multipDevc);
  hipMemcpy        (targetHost,targetDevc,Nround*sizeof(float4),hipMemcpyDeviceToHost);
// Direct summation on host
  float r;
  for( int i=0; i<N; i++ ) {
    float p = 0;
    for( int j=0; j<N; j++ ) {
      dx = targetHost[i].x - sourceHost[j].x;
      dy = targetHost[i].y - sourceHost[j].y;
      dz = targetHost[i].z - sourceHost[j].z;
      r = sqrtf(dx * dx + dy * dy + dz * dz + EPS2);
      p += sourceHost[j].w / r;
    }
    printf("%d %f %f\n",i,p,targetHost[i].w);
  }
}
