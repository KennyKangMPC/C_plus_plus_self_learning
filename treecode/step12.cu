
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

int   const THREADS = 8;
int   const NCRIT   = THREADS;
float const THETA   = 0.5;
float const EPS2    = 0.0001;

__device__ void multipole(int i, float4 &target, float *multipShrd) {
  float R, R3, R5;
  float3 d;
  d.x = target.x - multipShrd[i*13+0];
  d.y = target.y - multipShrd[i*13+1];
  d.z = target.z - multipShrd[i*13+2];
  R = rsqrtf(d.x * d.x + d.y * d.y + d.z * d.z);
  R3 = R * R * R;
  R5 = R3 * R * R;
  target.w += multipShrd[i*13+ 3] * R;
  target.w += multipShrd[i*13+ 4] * (-d.x * R3);
  target.w += multipShrd[i*13+ 5] * (-d.y * R3);
  target.w += multipShrd[i*13+ 6] * (-d.z * R3);
  target.w += multipShrd[i*13+ 7] * (3 * d.x * d.x * R5 - 1 * R3);
  target.w += multipShrd[i*13+ 8] * (3 * d.y * d.y * R5 - 1 * R3);
  target.w += multipShrd[i*13+ 9] * (3 * d.z * d.z * R5 - 1 * R3);
  target.w += multipShrd[i*13+10] * (3 * d.x * d.y * R5);
  target.w += multipShrd[i*13+11] * (3 * d.y * d.z * R5);
  target.w += multipShrd[i*13+12] * (3 * d.z * d.x * R5);
}

__global__ void kernel(int *offSrcGlob, float4 *sourceGlob, int *offMtpGlob, float *multipGlob, float4 *targetGlob) {
  int N = offSrcGlob[blockIdx.x+1]-offSrcGlob[blockIdx.x];
  int offset = offSrcGlob[blockIdx.x];
  float3 d;
  __shared__ float4 sourceShrd[THREADS];
  __shared__ float  multipShrd[13*THREADS];
  float4 target = targetGlob[blockIdx.x * THREADS + threadIdx.x];
  target.w *= -rsqrtf(EPS2);
  for( int iblok=0; iblok<(N-1)/THREADS; iblok++) {
    __syncthreads();
    sourceShrd[threadIdx.x] = sourceGlob[offset + iblok * THREADS + threadIdx.x];
    __syncthreads();
    for( int i=0; i<THREADS; i++ ) {
      d.x = target.x - sourceShrd[i].x;
      d.y = target.y - sourceShrd[i].y;
      d.z = target.z - sourceShrd[i].z;
      target.w += sourceShrd[i].w * rsqrtf(d.x * d.x + d.y * d.y + d.z * d.z + EPS2);
    }
  }
  int iblok = (N-1)/THREADS;
  __syncthreads();
  sourceShrd[threadIdx.x] = sourceGlob[offset + iblok * THREADS + threadIdx.x];
  __syncthreads();
  for( int i=0; i<N - (iblok * THREADS); i++ ) {
    d.x = target.x - sourceShrd[i].x;
    d.y = target.y - sourceShrd[i].y;
    d.z = target.z - sourceShrd[i].z;
    target.w += sourceShrd[i].w * rsqrtf(d.x * d.x + d.y * d.y + d.z * d.z + EPS2);
  }
  N = offMtpGlob[blockIdx.x+1]-offMtpGlob[blockIdx.x];
  offset = offMtpGlob[blockIdx.x];
  for( int iblok=0; iblok<(N-1)/THREADS; iblok++) {
    int index = offset + iblok * THREADS + threadIdx.x;
    __syncthreads();
    for( int i=0; i<13; i++ )
      multipShrd[threadIdx.x*13+i] = multipGlob[index*13+i];
    __syncthreads();
    for( int i=0; i<THREADS; i++ ) {
      multipole(i,target,multipShrd);
    }
  }
  iblok = (N-1)/THREADS;
  int index = offset + iblok * THREADS + threadIdx.x;
  __syncthreads();
  for( int i=0; i<13; i++ )
    multipShrd[threadIdx.x*13+i] = multipGlob[index*13+i];
  __syncthreads();
  for( int i=0; i<N - (iblok * THREADS); i++ ) {
    multipole(i,target,multipShrd);
  }
  targetGlob[blockIdx.x * THREADS + threadIdx.x] = target;
}

struct cell {
  int nleaf, nchild, leaf[NCRIT];
  float xc, yc, zc, r;
  float multipole[10];
  cell *parent, *child[8];
};

void initialize(cell *C) {
  C->nleaf = C->nchild = 0;
  C->parent = NULL;
  for( int i=0; i<8; i++ ) C->child[i] = NULL;
  for( int i=0; i<10; i++ ) C->multipole[i] = 0;
}

void add_child(int octant, cell *C, cell *&CN) {
  ++CN;
  initialize(CN);
  CN->r  = C->r/2;
  CN->xc = C->xc + CN->r * ((octant & 1) * 2 - 1);
  CN->yc = C->yc + CN->r * ((octant & 2) - 1    );
  CN->zc = C->zc + CN->r * ((octant & 4) / 2 - 1);
  CN->parent = C;
  C->child[octant] = CN;
  C->nchild |= (1 << octant);
}

void split_cell(float *x, float *y, float *z, cell *C, cell *&CN) {
  for( int i=0; i<NCRIT; i++ ) {                             
    int l = C->leaf[i];
    int octant = (x[l] > C->xc) + ((y[l] > C->yc) << 1) + ((z[l] > C->zc) << 2);
    if( !(C->nchild & (1 << octant)) ) add_child(octant,C,CN);
    cell *CC = C->child[octant];
    CC->leaf[CC->nleaf++] = l;
    if( CC->nleaf >= NCRIT ) split_cell(x,y,z,CC,CN);
  } 
}

void getMultipole(cell *C, float *x, float *y, float *z, float *m, cell **twig, int &ntwig) {
  float dx, dy, dz;
  if( C->nleaf >= NCRIT ) {
    for( int c=0; c<8; c++ )
      if( C->nchild & (1 << c) ) getMultipole(C->child[c],x,y,z,m,twig,ntwig);
  } else {
    for( int l=0; l<C->nleaf; l++ ) {
      int j = C->leaf[l];
      dx = C->xc - x[j];
      dy = C->yc - y[j];
      dz = C->zc - z[j];
      C->multipole[0] += m[j];
      C->multipole[1] += m[j] * dx;
      C->multipole[2] += m[j] * dy;
      C->multipole[3] += m[j] * dz;
      C->multipole[4] += m[j] * dx * dx / 2;
      C->multipole[5] += m[j] * dy * dy / 2;
      C->multipole[6] += m[j] * dz * dz / 2;
      C->multipole[7] += m[j] * dx * dy / 2;
      C->multipole[8] += m[j] * dy * dz / 2;
      C->multipole[9] += m[j] * dz * dx / 2;
    }
    twig[ntwig] = C;
    ntwig++;
  }
}

void upwardSweep(cell *C, cell *P) {
  float dx, dy, dz;
  dx = P->xc - C->xc;
  dy = P->yc - C->yc;
  dz = P->zc - C->zc;
  P->multipole[0] += C->multipole[0];
  P->multipole[1] += C->multipole[1] +  dx*C->multipole[0];
  P->multipole[2] += C->multipole[2] +  dy*C->multipole[0];
  P->multipole[3] += C->multipole[3] +  dz*C->multipole[0];
  P->multipole[4] += C->multipole[4] +  dx*C->multipole[1] + dx * dx * C->multipole[0] / 2;
  P->multipole[5] += C->multipole[5] +  dy*C->multipole[2] + dy * dy * C->multipole[0] / 2;
  P->multipole[6] += C->multipole[6] +  dz*C->multipole[3] + dz * dz * C->multipole[0] / 2;
  P->multipole[7] += C->multipole[7] + (dx*C->multipole[2] +      dy * C->multipole[1] + dx * dy * C->multipole[0]) / 2;
  P->multipole[8] += C->multipole[8] + (dy*C->multipole[3] +      dz * C->multipole[2] + dy * dz * C->multipole[0]) / 2;
  P->multipole[9] += C->multipole[9] + (dz*C->multipole[1] +      dx * C->multipole[3] + dz * dx * C->multipole[0]) / 2;
}

void evaluate(cell *CI, cell *CJ, float *x, float *y, float *z, float *m, float *p,
              int &offSrc, float4 *sourceHost, int &offMtp, float *multipHost) {
  float dx, dy, dz, r;
  if( CJ->nleaf >= NCRIT ) {
    for( int c=0; c<8; c++ ) {
      if( CJ->nchild & (1 << c) ) {
        cell *CC = CJ->child[c];
        dx = CI->xc - CC->xc;
        dy = CI->yc - CC->yc;
        dz = CI->zc - CC->zc;
        r = sqrtf(dx * dx + dy * dy + dz * dz);
        if( CI->r + CC->r > THETA*r ) {
          evaluate(CI,CC,x,y,z,m,p,offSrc,sourceHost,offMtp,multipHost);
        } else {
          multipHost[offMtp*13+ 0] = CC->xc;
          multipHost[offMtp*13+ 1] = CC->yc;
          multipHost[offMtp*13+ 2] = CC->zc;
          for( int i=0; i<10; i++ )
            multipHost[offMtp*13+ i + 3] = CC->multipole[i];
          offMtp++;
        }
      }
    }
  } else {
    for( int lj=0; lj<CJ->nleaf; lj++ ) {
      int j = CJ->leaf[lj];
      sourceHost[offSrc].x = x[j];
      sourceHost[offSrc].y = y[j];
      sourceHost[offSrc].z = z[j];
      sourceHost[offSrc].w = m[j];
      offSrc++;
    }
  }
}

int main() {
  int N = 50;
  float x[N],y[N],z[N],m[N],p[N],pd[N];
// Initialize
  for( int i=0; i<N; i++ ) {
    x[i] = rand() / (1. + RAND_MAX);
    y[i] = rand() / (1. + RAND_MAX);
    z[i] = rand() / (1. + RAND_MAX);
    m[i] = 1.0 / N;
  }
// Direct summation
  float dx, dy, dz, r;
  for( int i=0; i<N; i++ ) {
    float pp = - m[i] / sqrtf(EPS2);
    for( int j=0; j<N; j++ ) {
      dx = x[i] - x[j];
      dy = y[i] - y[j];
      dz = z[i] - z[j];
      r = sqrtf(dx * dx + dy * dy + dz * dz + EPS2);
      pp += m[j] / r;
    }
    pd[i] = pp;
  }

// Set root cell
  cell C0[N];
  initialize(C0);
  C0->xc = C0->yc = C0->zc = C0->r = 0.5;
// Build tree
  cell *CN = C0;
  for( int i=0; i<N; i++ ) {
    cell *C = C0;
    while( C->nleaf >= NCRIT ) {
      C->nleaf++;
      int octant = (x[i] > C->xc) + ((y[i] > C->yc) << 1) + ((z[i] > C->zc) << 2);
      if( !(C->nchild & (1 << octant)) ) add_child(octant,C,CN);
      C = C->child[octant];
    }
    C->leaf[C->nleaf++] = i;
    if( C->nleaf >= NCRIT ) split_cell(x,y,z,C,CN);
  }
// Multipole expansion
  int ntwig=0;
  cell *twig[N];
  getMultipole(C0,x,y,z,m,twig,ntwig);
// Upward translation
  for( cell *C=CN; C!=C0; --C ) {
    cell *P = C->parent;
    upwardSweep(C,P);
  }
// Evaluate expansion
  int Nround = ntwig * THREADS;
  int Nlist  = ntwig * Nround;
  int Mround =    13 * Nlist;
  int    *offSrcHost, *offSrcDevc;
  int    *offMtpHost, *offMtpDevc;
  float4 *sourceHost, *sourceDevc;
  float4 *targetHost, *targetDevc;
  float  *multipHost, *multipDevc;
// Allocate memory on host and device
  offSrcHost = (int   *)     malloc( (ntwig+1)*sizeof(int) );
  offMtpHost = (int   *)     malloc( (ntwig+1)*sizeof(int) );
  sourceHost = (float4*)     malloc(  Nlist*sizeof(float4) );
  targetHost = (float4*)     malloc( Nround*sizeof(float4) );
  multipHost = (float *)     malloc( Mround*sizeof(float ) );
  hipMalloc(  (void**) &offSrcDevc, (ntwig+1)*sizeof(int) );
  hipMalloc(  (void**) &offMtpDevc, (ntwig+1)*sizeof(int) );
  hipMalloc(  (void**) &sourceDevc,  Nlist*sizeof(float4) );
  hipMalloc(  (void**) &targetDevc, Nround*sizeof(float4) );
  hipMalloc(  (void**) &multipDevc, Mround*sizeof(float ) );
  for( int i=0; i<N; i++ ) p[i] = 0;
  int offSrc = 0, offMtp = 0;
  for( int t=0; t<ntwig; t++ ) {
    cell *CI = twig[t];
    cell *CJ = C0;
    offSrcHost[t] = offSrc;
    offMtpHost[t] = offMtp;
    for( int l=0; l<CI->nleaf; l++ ) {
      int i = CI->leaf[l];
      targetHost[t * THREADS + l].x = x[i];
      targetHost[t * THREADS + l].y = y[i];
      targetHost[t * THREADS + l].z = z[i];
      targetHost[t * THREADS + l].w = m[i];
    }
    evaluate(CI,CJ,x,y,z,m,p,offSrc,sourceHost,offMtp,multipHost);
  }
  offSrcHost[ntwig] = offSrc;
  offMtpHost[ntwig] = offMtp;
// Direct summation on device
  hipMemcpy(offSrcDevc,offSrcHost,(ntwig+1)*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(offMtpDevc,offMtpHost,(ntwig+1)*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(sourceDevc,sourceHost, Nlist*sizeof(float4),hipMemcpyHostToDevice);
  hipMemcpy(multipDevc,multipHost, Nlist*sizeof(float ),hipMemcpyHostToDevice);
  hipMemcpy(targetDevc,targetHost,Nround*sizeof(float4),hipMemcpyHostToDevice);
  kernel<<< Nround/THREADS, THREADS >>>(offSrcDevc,sourceDevc,offMtpDevc,multipDevc,targetDevc);
  hipMemcpy(targetHost,targetDevc,Nround*sizeof(float4),hipMemcpyDeviceToHost);
// Compare results
  float err=0, rel=0;
  for( int t=0; t<ntwig; t++ ) {
    cell *CI = twig[t];
    for( int l=0; l<CI->nleaf; l++ ) {
      int i = CI->leaf[l];
      p[i] += targetHost[t * THREADS + l].w;
      err += (pd[i] - p[i]) * (pd[i] - p[i]);
      rel += pd[i] * pd[i];
      printf("%d %f %f\n",i,pd[i],p[i]);
    }
  }
  printf("error : %f\n",sqrtf(err/rel));
}
