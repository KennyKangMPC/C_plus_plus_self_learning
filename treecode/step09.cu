
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

int   const N       = 33;
int   const THREADS = 8;
float const EPS2    = 0.0001;

__global__ void direct(float4 *sourceGlob, float *targetGlob) {
  float3 d;
  __shared__ float4 sourceShrd[THREADS];
  float4 target = sourceGlob[blockIdx.x * THREADS + threadIdx.x];
  target.w *= -rsqrtf(EPS2);
  for( int iblok=0; iblok<(N-1)/THREADS; iblok++) {
    __syncthreads();
    sourceShrd[threadIdx.x] = sourceGlob[iblok * THREADS + threadIdx.x];
    __syncthreads();
    for( int i=0; i<THREADS; i++ ) {
      d.x = target.x - sourceShrd[i].x;
      d.y = target.y - sourceShrd[i].y;
      d.z = target.z - sourceShrd[i].z;
      target.w += sourceShrd[i].w * rsqrtf(d.x * d.x + d.y * d.y + d.z * d.z + EPS2);
    }
  }
  int iblok = (N-1)/THREADS;
  __syncthreads();
  sourceShrd[threadIdx.x] = sourceGlob[iblok * THREADS + threadIdx.x];
  __syncthreads();
  for( int i=0; i<N - (iblok * THREADS); i++ ) {
    d.x = target.x - sourceShrd[i].x;
    d.y = target.y - sourceShrd[i].y;
    d.z = target.z - sourceShrd[i].z;
    target.w += sourceShrd[i].w * rsqrtf(d.x * d.x + d.y * d.y + d.z * d.z + EPS2);
  }
  targetGlob[blockIdx.x * THREADS + threadIdx.x] = target.w;
}

int main() {
  int Nround = (N / THREADS + 1) * THREADS;
  float4 *sourceHost, *sourceDevc;
  float  *targetHost, *targetDevc;
// Allocate memory on host and device
  sourceHost = (float4*)     malloc( Nround*sizeof(float4) );
  targetHost = (float *)     malloc( Nround*sizeof(float ) );
  hipMalloc(  (void**) &sourceDevc, Nround*sizeof(float4) );
  hipMalloc(  (void**) &targetDevc, Nround*sizeof(float ) );
// Initialize
  for( int i=0; i<N; i++ ) {
    sourceHost[i].x = rand() / (1. + RAND_MAX);
    sourceHost[i].y = rand() / (1. + RAND_MAX);
    sourceHost[i].z = rand() / (1. + RAND_MAX);
    sourceHost[i].w = 1.0 / N;
  }
// Direct summation on device
  hipMemcpy        (sourceDevc,sourceHost,Nround*sizeof(float4),hipMemcpyHostToDevice);
  direct<<< Nround/THREADS, THREADS >>>(sourceDevc,targetDevc);
  hipMemcpy        (targetHost,targetDevc,Nround*sizeof(float ),hipMemcpyDeviceToHost);
// Direct summation on host
  float dx, dy, dz, r;
  for( int i=0; i<N; i++ ) {
    float p = - sourceHost[i].w / sqrtf(EPS2);
    for( int j=0; j<N; j++ ) {
      dx = sourceHost[i].x - sourceHost[j].x;
      dy = sourceHost[i].y - sourceHost[j].y;
      dz = sourceHost[i].z - sourceHost[j].z;
      r = sqrtf(dx * dx + dy * dy + dz * dz + EPS2);
      p += sourceHost[j].w / r;
    }
    printf("%d %f %f\n",i,p,targetHost[i]);
  }
}
